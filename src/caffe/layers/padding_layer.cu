#include "hip/hip_runtime.h"

#include <vector>
#include "caffe/layers/padding_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PadImageConstNCHW(const int nthreads, const Dtype* const bottom_data, const int num,
    const int channels, const int height, const int width,
    const int padded_height, const int padded_width,
    const int pad_t, const int pad_l, Dtype value, Dtype* const top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        int nc = index / padded_width;
        const int pw = index % padded_width;
        const int ph = nc % padded_height;
        nc /= padded_height;
        const int h = ph - pad_t;
        const int w = pw - pad_l;
        top_data[index] = (h < 0 || w < 0 || h >= height || w >= width)
            ? value
            : bottom_data[(nc * height + h) * width + w];
  }
}

template <typename Dtype>
void PaddingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int count = top[0]->count();
    const int num = bottom[0]->shape(0);
    const int channels = bottom[0]->shape(1);
    const int height = bottom[0]->shape(2);
    const int width = bottom[0]->shape(3);

    const int padded_height = top[0]->shape(2);
    const int padded_width = top[0]->shape(3);

    PadImageConstNCHW<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count,
        bottom_data,
        num,
        channels,
        height,
        width,
        padded_height,
        padded_width,
        pad_t_,
        pad_l_,
        pad_value_,
        top_data);
}

template <typename Dtype>
void PaddingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    /* Not support */
}
INSTANTIATE_LAYER_GPU_FUNCS(PaddingLayer);
}



